
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20

const int kThreadsPerBlock = 256;

int nsteps,     // Number of time steps
    tpoints;    // Total points along string
float values[MAXPOINTS + 2];    // Values at time t

void check_param(void) {
    char tchar[20];

    // check number of points, number of iterations
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
        printf("Enter number of points along vibrating string [%d-%d]: ", MINPOINTS, MAXPOINTS);
        scanf("%s", tchar);
        tpoints = atoi(tchar);
        if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
            printf("Invalid. Please enter value between %d and %d\n", MINPOINTS, MAXPOINTS);
    }
    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
        printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
        scanf("%s", tchar);
        nsteps = atoi(tchar);
        if ((nsteps < 1) || (nsteps > MAXSTEPS))
            printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }
}

// Kernel for computing value of a point at specific time with speific time step
__global__ void init_and_update(float *dValues, int tpoints, int nsteps) {
}

void printfinal() {
    for (int i = 1; i <= tpoints; i++) {
        printf("%6.4f ", values[i]);
        if (i%10 == 0)
            printf("\n");
    }
}

int main(int argc, char *argv[]) {
    float *dValues; // Values in device
    int size = (MAXPOINTS + 2) * sizeof(float); // Size of memory to store values
    int numOfBlocks;    // Number of blocks used to call kernel

    sscanf(argv[1],"%d",&tpoints);
    sscanf(argv[2],"%d",&nsteps);
    check_param();
    hipMalloc(&dValues, size); // Allocate memory in device
    numOfBlocks = (tpoints - 1) / kThreadsPerBlock + 1; // Compute and ceil number of block
    init_and_update<<<numOfBlocks, kThreadsPerBlock>>>(dValues, tpoints, nsteps);
    hipMemcpy(values, dValues, size, hipMemcpyDeviceToHost);  // Copy result back to main memory
    printf("Printing final results...\n");
    printfinal();
    printf("\nDone.\n\n");
    hipFree(dValues);  // Free memory in device
    return 0;
}
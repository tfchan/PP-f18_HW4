
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20

int nsteps,     // Number of time steps
    tpoints;    // Total points along string
float values[MAXPOINTS + 2];    // Values at time t

void check_param(void) {
    char tchar[20];

    // check number of points, number of iterations
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
        printf("Enter number of points along vibrating string [%d-%d]: ", MINPOINTS, MAXPOINTS);
        scanf("%s", tchar);
        tpoints = atoi(tchar);
        if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
            printf("Invalid. Please enter value between %d and %d\n", MINPOINTS, MAXPOINTS);
    }
    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
        printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
        scanf("%s", tchar);
        nsteps = atoi(tchar);
        if ((nsteps < 1) || (nsteps > MAXSTEPS))
            printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }
}

void printfinal() {
    for (int i = 1; i <= tpoints; i++) {
        printf("%6.4f ", values[i]);
        if (i%10 == 0)
            printf("\n");
    }
}

int main(int argc, char *argv[]) {
    float *dValues; // Values in device
    int size = (MAXPOINTS + 2) * sizeof(float); // Size of memory to store values

    sscanf(argv[1],"%d",&tpoints);
    sscanf(argv[2],"%d",&nsteps);
    check_param();
    hipMalloc(&dValues, size);

    hipMemcpy(values, dValues, size, hipMemcpyDeviceToHost);
    printf("Printing final results...\n");
    printfinal();
    printf("\nDone.\n\n");
    hipFree(dValues);
    return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20

int nsteps,     // Number of time steps
    tpoints;    // Total points along string

void check_param(void) {
    char tchar[20];

    // check number of points, number of iterations
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
        printf("Enter number of points along vibrating string [%d-%d]: ", MINPOINTS, MAXPOINTS);
        scanf("%s", tchar);
        tpoints = atoi(tchar);
        if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
            printf("Invalid. Please enter value between %d and %d\n", MINPOINTS, MAXPOINTS);
    }
    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
        printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
        scanf("%s", tchar);
        nsteps = atoi(tchar);
        if ((nsteps < 1) || (nsteps > MAXSTEPS))
            printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }
}

int main(int argc, char *argv[]) {
    sscanf(argv[1],"%d",&tpoints);
    sscanf(argv[2],"%d",&nsteps);
    check_param();

    return 0;
}
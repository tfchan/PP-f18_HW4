
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

const int kThreadsPerBlock = 256;

int nsteps,     // Number of time steps
    tpoints;    // Total points along string
float values[MAXPOINTS + 2];    // Values at time t

void check_param(void) {
    char tchar[20];

    // check number of points, number of iterations
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
        printf("Enter number of points along vibrating string [%d-%d]: ", MINPOINTS, MAXPOINTS);
        scanf("%s", tchar);
        tpoints = atoi(tchar);
        if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
            printf("Invalid. Please enter value between %d and %d\n", MINPOINTS, MAXPOINTS);
    }
    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
        printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
        scanf("%s", tchar);
        nsteps = atoi(tchar);
        if ((nsteps < 1) || (nsteps > MAXSTEPS))
            printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }
    printf("Using points = %d, steps = %d\n", tpoints, nsteps);
}

// Device function to calculate new values using wave equation
__device__ float do_math(float value, float oldval) {
    float dtime = 0.3;
    float c = 1.0;
    float dx = 1.0;
    float tau = c * dtime / dx;
    float sqtau = tau * tau;
    return (2.0 * value - oldval + sqtau * (-2.0) * value);
}

// Kernel for computing value of a point at specific time with speific time step
__global__ void init_and_update(float *dValues, int tpoints, int nsteps) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float value, oldval, newval;
    if (index >= 1 && index <= tpoints) {
        // Initialization
        value = sin(2.0 * PI * (index - 1) / (tpoints - 1));
        // Update
        for (int i= 1; i <= nsteps; i++) {
            newval = (index == 1 || index == tpoints)? 0:do_math(value, oldval);
            oldval = value;
            value = newval;
        }
        dValues[index] = value;
    }
}

void printfinal() {
    for (int i = 1; i <= tpoints; i++) {
        printf("%6.4f ", values[i]);
        if (i%10 == 0)
            printf("\n");
    }
}

int main(int argc, char *argv[]) {
    float *dValues; // Values in device
    int size = (MAXPOINTS + 2) * sizeof(float); // Size of memory to store values
    int numOfBlocks;    // Number of blocks used to call kernel

    sscanf(argv[1],"%d",&tpoints);
    sscanf(argv[2],"%d",&nsteps);
    check_param();
    hipMalloc(&dValues, size); // Allocate memory in device
    numOfBlocks = (tpoints - 1) / kThreadsPerBlock + 1; // Compute and ceil number of block
    printf("Initializing points on the line...\n");
    printf("Updating all points for all time steps...\n");
    init_and_update<<<numOfBlocks, kThreadsPerBlock>>>(dValues, tpoints, nsteps);
    hipMemcpy(values, dValues, size, hipMemcpyDeviceToHost);  // Copy result back to main memory
    printf("Printing final results...\n");
    printfinal();
    printf("\nDone.\n\n");
    hipFree(dValues);  // Free memory in device
    return 0;
}